#include "hip/hip_runtime.h"
#include "generalIncludes.h"
#include "CudaRTXSolver.h"

__global__ void init(int width, int height, hiprandState* randStateuc, uc* out_image);
__global__ void createWorld(Object** objects, int num_ind, int num_vert, int* d_indeces, glm::vec3* d_verteces);
__global__ void cudaRTXSolveScene(uc* out_image, int m_x, int m_y, Camera* cam, Object** rs, hiprandState* randState, int format, int qual, int obj_num);
__global__ void addObject_to_scene(Object** scene_obj, int num_ind, int num_vert, int* d_indeces, glm::vec3* d_verteces);



const int t_x = 16, t_y = 16;
int obj_num = 0;
int width, height;
Object** d_objects;
hiprandState* d_randState;
dim3 blocks;
dim3 threads;
uc* out_image;

void RTX_Init(int _width, int _height, Object3D& obj) {
	width = _width;
	height = _height;

	hipDeviceSynchronize();
	hipDeviceReset();

	hipMallocManaged((void**)&out_image, width * height * 3 * sizeof(uc));

	hipMalloc((void**)&d_randState, width * height * sizeof(hiprandState));
	hipMalloc((void**)&d_objects, (obj.num_indeces / 3) * sizeof(Object*));

	glm::vec3* d_verteces;
	int* d_indeces;
	hipMalloc((void**)&d_indeces, sizeof(int) * obj.num_indeces);

	hipMalloc((void**)&d_verteces, sizeof(glm::vec3) * obj.num_vert);
	hipMemcpy(d_indeces, obj.indeces, sizeof(int) * obj.num_indeces, hipMemcpyHostToDevice);
	hipMemcpy(d_verteces, obj.verteces, sizeof(glm::vec3) * obj.num_vert, hipMemcpyHostToDevice);
	obj_num = obj.num_indeces / 3;

	createWorld << <1, 1 >> > (d_objects, obj.num_indeces, obj.num_vert, d_indeces, d_verteces);
	hipDeviceSynchronize();

	blocks = dim3(width / t_x + 1, height / t_y + 1);
	threads = dim3(t_x, t_y);

	init << < blocks, threads >> > (width, height, d_randState, out_image); //����� �����... �� ��������, ���� t_x = t_y < 32
	hipDeviceSynchronize();
}

uc* RTXSolveScene(Camera& camera, int format, int qual) {
	Camera* d_camera;

	hipMalloc((void**)&d_camera, sizeof(Camera));
	hipMemcpy(d_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	cudaRTXSolveScene << <blocks, threads >> > (out_image, width, height, d_camera, d_objects, d_randState, format, qual, obj_num);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float tm;
	hipEventElapsedTime(&tm, start, stop);
	std::cout << tm << " ms / fps " << 1000.f / tm << '\n';
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipDeviceSynchronize();

	hipFree(d_camera);

	return out_image;
}
void host_addObjectToScene(Object** scene_obj, int num_ind, int num_vert, int* d_indeces, glm::vec3* d_verteces) {
	addObject_to_scene << <1, 1 >> > (scene_obj, num_ind, num_vert, d_indeces, d_verteces);
}
#define RANDVEC3  glm::vec3(hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state))

__device__ glm::vec3 random_in_unit_sphere(hiprandState* local_rand_state) {
	glm::vec3 p;
	p = 2.0f * RANDVEC3 - glm::vec3(1);
	return glm::normalize(p);
}

__device__  glm::vec3 color(const ray& r, Object** world, hiprandState* local_rand_state, int obj_num) {
	ray cur_ray = r;
	float cur_attenuation = 1.0f;
	Object* last_hitted = nullptr;
	Material curMat;
	glm::vec3 end_color;
	float n[5] = { 1,1,1,1,1 };
	int n_p = 0;

	for (int i = 0; i < 10; i++) {
		hitData rec(false);
		double min_dist = 1.79769e+308;
		for (int i = 0; i < obj_num; i++) {
			if (*(world + i) != last_hitted) {
				hitData cur_data = (*(world + i))->hit(cur_ray);
				if (min_dist > cur_data.h_distance && cur_data.isHit) {
					min_dist = cur_data.h_distance;
					last_hitted = *(world + i);
					rec = cur_data;
				}
			}
		}
		
		if (rec.isHit){//����
			if (cur_attenuation == 1.0f) {
				curMat = *(last_hitted->mat);
				end_color = (*last_hitted->mat).color;
			}
			
			if (last_hitted->mat->type == 0) {
				cur_attenuation *= 0.5f;
				cur_ray = ray(rec.h_point + glm::vec3(0.001f) * rec.h_normal,
					rec.h_point + glm::vec3(1.0f) * rec.h_normal + glm::vec3(1.0f) * random_in_unit_sphere(local_rand_state));
			}
			else if (last_hitted->mat->type == 1) {
				cur_attenuation *= 0.5f;
				glm::vec3 dir = glm::normalize(cur_ray.to - cur_ray.from);
				cur_ray = ray(rec.h_point + glm::vec3(0.001f) * rec.h_normal,
					rec.h_point + glm::vec3(0.001f) * rec.h_normal + glm::normalize(-glm::vec3(2.f) * glm::dot(dir, rec.h_normal) * rec.h_normal + dir));
			}
			else if (last_hitted->mat->type == 2) {
				glm::vec3 dir = glm::normalize(cur_ray.to - cur_ray.from);
				float cs = glm::dot(dir, rec.h_normal);
				if (cs < 0) {
					float ang = glm::acos(cs);
					float sin2 = (n[n_p] / last_hitted->mat->n) * glm::sin(ang);
					
					if (sin2 < 1) {

						n_p++;
						n[n_p] = last_hitted->mat->n;
						float ang2 = glm::asin(sin2);
						cur_ray = ray(rec.h_point - glm::vec3(0.001f) * rec.h_normal,
							rec.h_point - glm::vec3(0.001f) * rec.h_normal + glm::normalize(glm::rotate(glm::vec3(-1) * rec.h_normal, -ang2, glm::cross(rec.h_normal, dir))));
					}
					else {
						cur_ray = ray(rec.h_point - glm::vec3(0.001f) * rec.h_normal,
							rec.h_point - glm::vec3(0.001f) * rec.h_normal + glm::normalize(glm::dot(dir, glm::vec3(-1) * rec.h_normal) * glm::vec3(-1) * rec.h_normal + dir));
					}
				}
				else {
					float ang = glm::acos(cs);
					float sin2 = (n[n_p] / n[n_p - 1]) * glm::sin(ang);
					
					if (sin2 < 1) {
						cur_attenuation *= 0.8f;
						n[n_p] = 1;
						n_p--;
						float ang2 = glm::asin(sin2);
						cur_ray = ray(rec.h_point + glm::vec3(0.001f) * rec.h_normal,
							rec.h_point + glm::vec3(0.001f) * rec.h_normal + glm::normalize(glm::rotate(rec.h_normal, ang2, glm::cross(rec.h_normal, dir))));
					}
					else {
						cur_ray = ray(rec.h_point + glm::vec3(0.001f) * rec.h_normal,
							rec.h_point + glm::vec3(0.001f) * rec.h_normal + glm::normalize(-glm::vec3(2.f) * glm::dot(dir, rec.h_normal) * rec.h_normal + dir));
					}
				}
				
			}
			if (curMat.type == 1) {
				end_color = (end_color + (*last_hitted->mat).color * cur_attenuation) / 2.f;
			}
		}
		else {
			glm::vec3 unit_direction = glm::normalize(cur_ray.to - cur_ray.from);
			float t = 0.5f * (unit_direction.y + 1.0f);
			glm::vec3 c = (1.0f - t) * glm::vec3(1.0, 1.0, 1.0) + t * glm::vec3(0.5, 0.7, 1.0);
			if (cur_attenuation != 1)
				return cur_attenuation * end_color;
			else return glm::vec3(1, 1, 1);
		}
	}
	return cur_attenuation * end_color;
}

__global__ void init(int width, int height, hiprandState* randState, uc* out_image) {
	int _x = threadIdx.x + blockDim.x * blockIdx.x,
		_y = threadIdx.y + blockDim.y * blockIdx.y;
	if ((_x >= width) || (_y >= height)) return;
	int pixel_id = _y * width + _x;
	hiprand_init(1337, pixel_id, 0, &randState[pixel_id]);
	out_image[pixel_id + 0] = 0;
	out_image[pixel_id + 1] = 0;
	out_image[pixel_id + 2] = 0;
}

__global__ void createWorld(Object** objects, int num_ind, int num_vert, int* d_indeces, glm::vec3* d_verteces) {
	 if (threadIdx.x == 0 && blockIdx.x == 0) {

		for (int i = 0; i < num_ind / 3; i++) {
		 *(objects + i) = new renderTriangle(d_verteces[d_indeces[i * 3] - 1], d_verteces[d_indeces[i * 3 + 1] - 1], d_verteces[d_indeces[i * 3 + 2] - 1]);
		 (*(objects + i))->mat = new Diffuse(glm::vec3(0.8, 0.8, 0.8));
		}
		
		//*(objects + 0) = new renderTriangle(glm::vec3(-5, 0, -5), glm::vec3(-5, 0, 5), glm::vec3(5, 0, 5) );
		//(*(objects + 0))->mat = new Diffuse(glm::vec3(1,1,1));
		//*(objects + 1) = new renderTriangle(glm::vec3(5, 0, -5), glm::vec3(-5, 0, -5), glm::vec3(5, 0, 5) );
		//(*(objects + 1))->mat =new Diffuse(glm::vec3(1,1,1));
		//*(objects + 2) = new renderSphere(3);
		//(*(objects + 2))->pos = glm::vec3(0,3,0);
		//(*(objects + 2))->mat = new Optical(1.01f);

	 }
}

__global__ void cudaRTXSolveScene(uc* out_image, int m_x, int m_y, Camera* cam, Object** rs, hiprandState* randState, int format, int qual, int obj_num) {
	int _x = threadIdx.x + blockDim.x * blockIdx.x,
		_y = threadIdx.y + blockDim.y * blockIdx.y;
	if ((_x >= m_x) || (_y >= m_y)) return;
	
	// ������ ���� � �������
	glm::vec3 cam_right = glm::cross(cam->cam_front, cam->cam_up);
	double sc_u = (2 * cam->min_dist * tan((cam->fov / 2.f) / 180.f * pi)) / m_x;
	ray r;
	r.from = cam->cam_pos;
	r.to = cam->cam_pos + cam->min_dist * cam->cam_front + 
			(float)(sc_u * ((-m_x / 2) + _x))* cam_right + 
			(float)(sc_u * ((m_y / 2) - _y)) * cam->cam_up;
	//
	int pix_id = (_y * m_x + _x) * 3;

	hiprandState local_rand_state = randState[pix_id / 3];

	// ������ ����

	if (format == RTX_RENDER_COLOR) {
		glm::vec3 col(0, 0, 0);
		for (int s = 0; s < qual; s++) {
			//printf("%d\n", (int)100 * ((float)s / qual));
			col += color(r, rs, &local_rand_state, obj_num);
		}
		col /= float(qual);
		col.x = sqrt(col.x);
		col.y = sqrt(col.y);
		col.z = sqrt(col.z);
		out_image[pix_id + 0] = 255 * col.x;
		out_image[pix_id + 1] = 255 * col.y;
		out_image[pix_id + 2] = 255 * col.z;
	}
	else if (format == RTX_RENDER_NORMALS) {
		hitData rec;
		double min_dist = 1.79769e+308;
		clock_t a, b;

		
		for (int i = 0; i < obj_num; i++) {
			hitData cur_data = (*(rs + i))->hit(r);
			if (min_dist > cur_data.h_distance && cur_data.isHit) {
				min_dist = cur_data.h_distance;
				rec = cur_data;
			}
		}

		if (rec.isHit) {
			out_image[pix_id + 0] = 127.5f * (1 + rec.h_normal.x);
			out_image[pix_id + 1] = 127.5f * (1 + rec.h_normal.y);
			out_image[pix_id + 2] = 127.5f * (1 + rec.h_normal.z);
		}
		else {
			out_image[pix_id + 0] = 255;
			out_image[pix_id + 1] = 255;
			out_image[pix_id + 2] = 255;
		}
	}

	//
}

__global__ void addObject_to_scene(Object** scene_obj, int num_ind, int num_vert, int* d_indeces, glm::vec3* d_verteces) {
	
}